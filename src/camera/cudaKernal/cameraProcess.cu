
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


__global__ static void greyRgbFlatArray(unsigned char *arrIn, unsigned char *arrOut, int imgPoints)
{
    int pointIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (3 * pointIndex < imgPoints)
    {
        unsigned char grey = 0.299f*arrIn[3*pointIndex] +  0.587f *arrIn[3*pointIndex + 1] + 0.114f *arrIn[3*pointIndex + 2];
        arrOut[pointIndex] = grey;
        arrOut[pointIndex + 1] = grey;
        arrOut[pointIndex + 2] = grey;
    }
}

void rgbFlatGrey(unsigned char* devPtr, unsigned char* HostIn, unsigned char* HostOut, int imgPoints)
{
    hipMemcpy(devPtr, HostIn, 3 * imgPoints * sizeof(unsigned char), hipMemcpyHostToDevice);
    greyRgbFlatArray<<<1, 1024>>>(devPtr, devPtr, imgPoints);
    hipMemcpy(HostOut, devPtr, 3 * imgPoints * sizeof(unsigned char), hipMemcpyDeviceToHost);
}
